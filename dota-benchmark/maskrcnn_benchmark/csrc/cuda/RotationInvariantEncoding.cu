#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

typedef unsigned long uint64;
typedef unsigned int uint32;
typedef unsigned short uint16;
typedef unsigned char uint8;

#define _FLT_MAX 3.402823466e+38F

#define CUDA_KERNEL_LOOP(i, n) \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N){
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void AlignFeatureKernel(
    const uint32 nthreads, 
    const torch::PackedTensorAccessor<Dtype, 4, torch::RestrictPtrTraits, size_t> feature_data,
    const uint16 nBatch,
    const uint16 nFeature,
    const uint8 H,
    const uint8 W,
    const uint8 nOrientation,
    torch::PackedTensorAccessor<uint8, 4, torch::RestrictPtrTraits, size_t> mainDirection_data,
    torch::PackedTensorAccessor<Dtype, 4, torch::RestrictPtrTraits, size_t> aligned_data)
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        uint8 l, h, w;
        
        const uint16 j = n % nFeature;
        const uint16 i = n / nFeature;
        
        for(h = 0; h < H; h++){
        	for(w = 0; w < W; w++){
        		Dtype maxVal = -_FLT_MAX;
				for (l = 0; l < nOrientation; l++) {
					Dtype val = feature_data[i][j * nOrientation + l][h][w];
					if (val > maxVal) {
						maxVal = val;
						mainDirection_data[i][j][h][w] = l;
					}
				}
				
		        for (l = 0; l < nOrientation; l++) {
		        	Dtype src = feature_data[i][j * nOrientation + l][h][w];
		            uint8 alignedIndex = (l - mainDirection_data[i][j][h][w] + nOrientation) % nOrientation;
		            aligned_data[i][j * nOrientation + alignedIndex][h][w] = src;
		        }
        	}
        }
    }
}

std::vector<torch::Tensor> RIE_AlignFeature_forward_cuda(
    const torch::Tensor feature,
    const uint8 nOrientation)
{
    AT_ASSERTM(feature.type().is_cuda(), "feature must be a CUDA tensor");
//    AT_ASSERTM((feature.size(2) == 1) and (feature.size(3) == 1), "feature must be 1-D tensor in dim=2, 3");
    
    const uint16 nBatch = feature.size(0);
    const uint16 nChannel = feature.size(1);
    const uint8 H = feature.size(2);
    const uint8 W = feature.size(3);
    const uint16 nFeature = nChannel / nOrientation;
    const uint32 count = nBatch * nFeature;
    
    const auto feature_data = feature;
    auto mainDirection_data = torch::zeros({nBatch, nFeature, H, W}, feature.options().dtype(at::kByte).device(at::kCUDA));
    auto aligned_data = torch::zeros({nBatch, nChannel, H, W}, feature.options().dtype(at::kFloat).device(at::kCUDA));

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    AT_DISPATCH_FLOATING_TYPES(feature.type(), "rie_cuda_forward", [&] {
    	AlignFeatureKernel<scalar_t> <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream >>>(
                count,
                feature_data.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
                nBatch,
                nFeature,
                H,
                W,
                nOrientation,
                mainDirection_data.packed_accessor<uint8, 4, torch::RestrictPtrTraits, size_t>(),
    			aligned_data.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>());
    });
        
	THCudaCheck(hipGetLastError());
	return {aligned_data, mainDirection_data};
}

template <typename Dtype>
__global__ void UnAlignFeatureKernel(
    const uint32 nthreads,
    const torch::PackedTensorAccessor<Dtype, 4, torch::RestrictPtrTraits, size_t> feature_data,
    const uint16 nBatch,
    const uint16 nFeature,
    const uint8 H,
    const uint8 W,
    const uint8 nOrientation,
    torch::PackedTensorAccessor<uint8, 4, torch::RestrictPtrTraits, size_t> mainDirection_data,
    torch::PackedTensorAccessor<Dtype, 4, torch::RestrictPtrTraits, size_t> unaligned_data)
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        uint8 l, w, h;
        
        const uint16 j = n % nFeature;
        const uint16 i = n / nFeature;
        for(h = 0; h < H; h++){
        	for(w = 0; w < W; w++){
        		for (l = 0; l < nOrientation; l++) {
        		        	Dtype src = feature_data[i][j * nOrientation + l][h][w];
        		            uint8 alignedIndex = (l + mainDirection_data[i][j][h][w]) % nOrientation;
        		            unaligned_data[i][j * nOrientation + alignedIndex][h][w] = src;
				}
        	}
        }
        
    }
}

torch::Tensor RIE_AlignFeature_backward_cuda(
    const torch::Tensor feature,	//feature is the align output grad paras
    const torch::Tensor mainDirection,
    const uint8 nOrientation)
{
    AT_ASSERTM(feature.type().is_cuda(), "feature must be a CUDA tensor");
//    AT_ASSERTM((feature.size(2) == 1) and (feature.size(3) == 1), "feature must be 1-D tensor in dim=2, 3");

    const uint16 nBatch = feature.size(0);
    const uint16 nChannel = feature.size(1);
    const uint8 H = feature.size(2);
	const uint8 W = feature.size(3);
    const uint16 nFeature = nChannel / nOrientation;
    const uint32 count = nBatch * nFeature;

    const auto feature_data = feature;		
    const auto mainDirection_data = mainDirection;
    auto unaligned_data = torch::zeros({nBatch, nChannel, H, W}, feature.options().dtype(at::kFloat).device(at::kCUDA));
    //feature is the align output grad paras
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES(feature.type(), "rie_cuda_backward", [&] {
    	UnAlignFeatureKernel<scalar_t> <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream >>>(
                count,
                feature_data.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>(),
                nBatch,
                nFeature,
                H,
                W,
                nOrientation,
                mainDirection_data.packed_accessor<uint8, 4, torch::RestrictPtrTraits, size_t>(),
    			unaligned_data.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits, size_t>());
    });

	THCudaCheck(hipGetLastError());
	return unaligned_data;
}